#include "hip/hip_runtime.h"
#include <iostream>
#include "checkers.cuh"
#include "initial.cuh"

int main() {
  int *A = initial_state();
  my_rep_class *my_rep;
  int *n_moves;
  int4 *moves;

  hipMalloc(&my_rep, sizeof(my_rep_class));
  hipMalloc(&n_moves, sizeof(int));
  hipMalloc(&moves, 1024*2048*4*sizeof(int4));
  my_representation<<<my_representation_blocks,
    my_representation_threads>>>(A, my_rep);
  for (int i = 0; i < 100; i++){
    hipMemset(n_moves, 0, sizeof(int));
    possible_moves<<<possible_moves_blocks,
      possible_moves_threads>>>(my_rep, 1, n_moves, moves);
    update<<<update_blocks, update_threads>>>(my_rep,
      selected_move(n_moves, moves));
    hipMemset(n_moves, 0, sizeof(int));
    possible_moves<<<possible_moves_blocks,
      possible_moves_threads>>>(my_rep, 2, n_moves, moves);
    update<<<update_blocks, update_threads>>>(my_rep,
      selected_move(n_moves, moves));
  }

  hipFree(n_moves);
  hipFree(moves);
  hipFree(my_rep);
  hipFree(A);
  // free(A);
  // delete[] A;
  return 0;
}
