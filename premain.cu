#include "hip/hip_runtime.h"
#include <iostream>
#include "checkers.cuh"
#include "initial.cuh"

int main() {
  int *A = initial_state();
  my_rep_class *my_rep;
  int *n_moves;
  int4 *moves;

  // for (int i = 0; i < 20; ++i) {
  //   std::cout << i << ": " << A[i] << "\n";
  // }

  hipMalloc(&my_rep, sizeof(my_rep_class));
  hipMalloc(&n_moves, sizeof(int));
  hipMalloc(&moves, 1024*2048*4*sizeof(int4));
  my_representation<<<my_representation_blocks,
    my_representation_threads>>>(A, my_rep);


  hipFree(n_moves);
  hipFree(moves);
  hipFree(my_rep);
  hipFree(A);
  // free(A);
  // delete[] A;
  return 0;
}
