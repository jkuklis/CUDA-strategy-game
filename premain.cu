#include "hip/hip_runtime.h"
#include <iostream>
#include "checkers.cuh"
#include "initial.cuh"

int main() {
  hipFree(0);

  int *A = initial_state();
  my_rep_class *my_rep;
  int *n_moves;
  int4 *moves;

  hipMalloc(&my_rep, sizeof(my_rep_class));
  hipMalloc(&n_moves, sizeof(int));
  hipMalloc(&moves, 1024*2048*4*sizeof(int4));
  // hipMalloc(&moves, 4*8*4*sizeof(int4));
  // my_representation<<<my_representation_blocks,
  //   my_representation_threads>>>(A, my_rep);

  // std::cout << "\n";
  // for (int i = 0; i < 10; i++) {
  //   for (int j = 0; j < 10; j++) {
  //     int a;
  //     hipMemcpy(&a, &(my_rep->A[i * 10 + j]), sizeof(int), hipMemcpyDeviceToHost);
  //     if (a != 1) {
  //       std::cout << a << " ";
  //     } else {
  //       std::cout << "  ";
  //     }
  //   }
  //   std::cout << "\n";
  // }
  //
  // int turns = 1;
  //
  // for (int i = 0; i < turns; i++){
  //   hipMemset(n_moves, 0, sizeof(int));
  //   possible_moves<<<possible_moves_blocks,
  //     possible_moves_threads>>>(my_rep, 1, n_moves, moves);
  //   update<<<update_blocks, update_threads>>>(my_rep,
  //     selected_move(n_moves, moves));
  //
  //   // std::cout << "\n";
  //   // int nm;
  //   // hipMemcpy(&nm, n_moves, sizeof(int), hipMemcpyDeviceToHost);
  //   // for (int i = 0; i < nm; i++) {
  //   //   int4 a;
  //   //   hipMemcpy(&a, &moves[i], sizeof(int4), hipMemcpyDeviceToHost);
  //   //   std::cout << a.x << " " << a.y << " " << a.z << " " << a.w << "\n";
  //   // }
  //   // std::cout << "\n";
  //   // for (int i = 0; i < 10; i++) {
  //   //   for (int j = 0; j < 10; j++) {
  //   //     int a;
  //   //     hipMemcpy(&a, &(my_rep->A[i * 10 + j]), sizeof(int), hipMemcpyDeviceToHost);
  //   //     if (a != 1) {
  //   //       std::cout << a << " ";
  //   //     } else {
  //   //       std::cout << "  ";
  //   //     }
  //   //   }
  //   //   std::cout << "\n";
  //   // }
  //
  //   hipMemset(n_moves, 0, sizeof(int));
  //   possible_moves<<<possible_moves_blocks,
  //     possible_moves_threads>>>(my_rep, 2, n_moves, moves);
  //   update<<<update_blocks, update_threads>>>(my_rep,
  //     selected_move(n_moves, moves));
  //
  //   // hipDeviceSynchronize();
  //
  //   // std::cout << "\n";
  //   // hipMemcpy(&nm, n_moves, sizeof(int), hipMemcpyDeviceToHost);
  //   // for (int i = 0; i < nm; i++) {
  //   //   int4 a;
  //   //   hipMemcpy(&a, &moves[i], sizeof(int4), hipMemcpyDeviceToHost);
  //   //   std::cout << a.x << " " << a.y << " " << a.z << " " << a.w << "\n";
  //   // }
  //   // std::cout << "\n";
  //   // for (int i = 0; i < 10; i++) {
  //   //   for (int j = 0; j < 10; j++) {
  //   //     int a;
  //   //     hipMemcpy(&a, &(my_rep->A[i * 10 + j]), sizeof(int), hipMemcpyDeviceToHost);
  //   //     if (a != 1) {
  //   //       std::cout << a << " ";
  //   //     } else {
  //   //       std::cout << "  ";
  //   //     }
  //   //   }
  //   //   std::cout << "\n";
  //   // }
  // }

  // int side = 2 + 248;
  //
  // std::cout << "\n";
  // for (int i = 0; i < side; i++) {
  //   for (int j = 0; j < side; j++) {
  //     int a;
  //     hipMemcpy(&a, &(my_rep->A[i * side + j]), sizeof(int), hipMemcpyDeviceToHost);
  //     if (a != 1) {
  //       std::cout << a << " ";
  //     } else {
  //       std::cout << "  ";
  //     }
  //   }
  //   std::cout << "\n";
  // }

  hipFree(n_moves);
  hipFree(moves);
  hipFree(my_rep);
  hipFree(A);
  // free(A);
  // delete[] A;
  return 0;
}
